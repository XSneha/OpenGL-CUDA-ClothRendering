﻿//"C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Auxiliary\Build\vcvars64.bat"

#include "CUDAClothCommonGlobal.h"

bool gbFullscree = false;
bool gbActiveWindow = false;
WINDOWPLACEMENT wpPrev = { sizeof(WINDOWPLACEMENT) };

HDC ghdc = NULL;
HGLRC ghrc = NULL;

bool bOnGPU = false;

DWORD dwStyle;
HWND ghwnd;

FILE* gpFile;

int WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR cmdLine, int iCmdShow) {

	void Initialize(void);
	void Display(void);

	WNDCLASSEX wndclass;
	MSG msg;
	HWND hwnd;
	TCHAR szAppName[] = TEXT("OpenGl - CUDA Interop Template");
	bool bDone = false;

	if (fopen_s(&gpFile, "MyLog.txt", "w") != 0) {
		MessageBox(NULL, TEXT("Failed to Open file Mylog.txt"), TEXT("ERROR"), MB_OK);
		return (0);
	}

	wndclass.cbSize = sizeof(WNDCLASSEX);
	wndclass.lpszClassName = szAppName;
	wndclass.style = CS_HREDRAW | CS_VREDRAW | CS_OWNDC;
	wndclass.lpfnWndProc = WndProc;
	wndclass.hInstance = hInstance;
	wndclass.hbrBackground = (HBRUSH)GetStockObject(WHITE_BRUSH);
	wndclass.hCursor = LoadCursor(NULL, IDC_ARROW);
	wndclass.hIcon = LoadIcon(hInstance, NULL);
	wndclass.hIconSm = LoadIcon(hInstance, NULL);
	wndclass.cbClsExtra = 0;
	wndclass.cbWndExtra = 0;
	wndclass.lpszMenuName = NULL;

	RegisterClassEx(&wndclass);

	hwnd = CreateWindowEx(WS_EX_APPWINDOW,
		szAppName,
		TEXT("Cloth Rendering."),
		WS_OVERLAPPEDWINDOW | WS_CLIPCHILDREN | WS_VISIBLE,
		100,
		100,
		WIN_WIDTH,
		WIN_HEIGHT,
		0,
		0,
		hInstance,
		NULL);

	if (hwnd == NULL) {
		MessageBox(NULL, TEXT("Failed to Create Window."), TEXT("ERROR!"), MB_OK);
		exit(0);
	}
	ghwnd = hwnd;

	Initialize();
	ShowWindow(hwnd, iCmdShow);

	SetForegroundWindow(hwnd);
	SetFocus(hwnd);

	while (bDone == false) {
		if (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE)) {
			if (msg.message == WM_QUIT) {
				bDone = true;
			}
			else {
				TranslateMessage(&msg);
				DispatchMessage(&msg);
			}
		}
		else {
			if (gbActiveWindow == true) {
				Display();
			}
		}
	}

	return (int)msg.wParam;
}

LRESULT CALLBACK WndProc(HWND hwnd, UINT iMsg, WPARAM wParam, LPARAM lParam) {

	void Resize(int, int);
	void UnInitialize(void);
	void ToggleFullscreen(void);

	MONITORINFO mi = { sizeof(MONITORINFO) };

	switch (iMsg) {
	case WM_SETFOCUS:
		gbActiveWindow = true;
		break;
	case WM_KILLFOCUS:
		gbActiveWindow = false;
		break;
	case WM_SIZE:
		Resize(LOWORD(lParam), HIWORD(lParam));
		break;
	case WM_ERASEBKGND:
		return(0);
	case WM_KEYDOWN:
		switch (wParam) {
		case VK_ESCAPE:DestroyWindow(ghwnd);
			break;
		case 0x46:
		case 0x66:
			ToggleFullscreen();
			break;
		case 'C':
		case 'c':
			bOnGPU = false;
			break;
		case 'G':
		case 'g':
			bOnGPU = true;
			break;
		default:
			break;
		}
		break;
	case WM_DESTROY:
		UnInitialize();
		PostQuitMessage(0);
		break;
	default:break;
	}

	return DefWindowProc(hwnd, iMsg, wParam, lParam);
}

void ToggleFullscreen(void) {
	MONITORINFO mi = { sizeof(MONITORINFO) };
	if (gbFullscree == false) {
		dwStyle = GetWindowLong(ghwnd, GWL_STYLE);
		if (dwStyle & WS_OVERLAPPEDWINDOW) {
			if (GetWindowPlacement(ghwnd, &wpPrev) && GetMonitorInfo(MonitorFromWindow(ghwnd, MONITORINFOF_PRIMARY), &mi)) {
				SetWindowLong(ghwnd, GWL_STYLE, dwStyle & ~WS_OVERLAPPEDWINDOW);
				SetWindowPos(ghwnd,
					HWND_TOP,
					mi.rcMonitor.left,
					mi.rcMonitor.top,
					mi.rcMonitor.right - mi.rcMonitor.left,
					mi.rcMonitor.bottom - mi.rcMonitor.top,
					SWP_NOZORDER | SWP_FRAMECHANGED);
			}
		}
		ShowCursor(FALSE);
		gbFullscree = true;
	}
	else {
		ShowCursor(TRUE);
		SetWindowLong(ghwnd, GWL_STYLE, dwStyle);
		SetWindowPlacement(ghwnd, &wpPrev);
		SetWindowPos(ghwnd, HWND_TOP, 0, 0, 0, 0, SWP_NOMOVE | SWP_NOSIZE | SWP_NOZORDER | SWP_FRAMECHANGED | SWP_NOOWNERZORDER);
		gbFullscree = false;
	}
}


void Resize(int width, int height) {
	if (height == 0)
		height = 1;
	glViewport(0, 0, (GLsizei)width, (GLsizei)height);

	//gluPerspective(45.0f, (GLfloat)width / (GLfloat)height, 0.1f, 100.0f);
	perspectiveProjectionMatrix = vmath::perspective(45.0f, (GLfloat)width / (GLfloat)height, 0.1f, 100.0f);
}

